#include "hip/hip_runtime.h"
/*
============================================================================
Name        : Senior.c
Author      :
Version     :
Copyright   : Your copyright notice
Description : Hello World in C, Ansi-style
============================================================================
*/
#include <time.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "Las.h"
#include <iostream>
#include <fstream>
#include <hip/hip_complex.h>
#include <complex>

#define X 0
#define Y 1
#define Z 2
using namespace std;
typedef struct Qbits{
	hipDoubleComplex  i[2];
}Qbit;
hipDoubleComplex cuCsin(hipDoubleComplex x){
	double real = hipCreal(x);
	double imag = hipCimag(x);

	return make_hipDoubleComplex(sin(real)*cosh(imag), cos(real)*sinh(imag));

}
__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
	double factor = exp(x.x);
	return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}
hipDoubleComplex cuCcos(hipDoubleComplex x){
	double real = hipCreal(x);
	double imag = hipCimag(x);

	return make_hipDoubleComplex(cos(real)*cosh(imag), -sin(real)*sinh(imag));

}
void printm(hipDoubleComplex *C, int nrows, int ncols){
	int i;
	FILE * pFile;
	pFile = fopen("myfiles.txt", "w");
	for (i = 0; i<ncols*nrows; i++){
		if (i % (ncols) == 0)fprintf(pFile,"\n");
		fprintf(pFile, "%2lf + I * %2lf ", hipCreal(*(C + i)), hipCimag(*(C + i)));
	}
	fprintf(pFile, "\n");
	fclose(pFile);
}
typedef struct Gates
{
	int index1;
	int index2;
	int direction;
	hipDoubleComplex parameter;
} Gate;
void generateRotationalMatrix(Gate g, hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex *C, int index, int numberOfWires )
{
	switch (g.direction){
	case X:
		;
		//Ensure whether it is /2 or not
		A[0] = cuCcos(hipCdiv(g.parameter, make_hipDoubleComplex(2, 0)));
		A[1] = hipCmul(make_hipDoubleComplex(0, -1), cuCsin(hipCdiv(g.parameter, make_hipDoubleComplex(2, 0))));
		A[2] = hipCmul(make_hipDoubleComplex(0, -1), hipCdiv(g.parameter, make_hipDoubleComplex(2, 0)));
		A[3] = cuCcos(hipCdiv(g.parameter, make_hipDoubleComplex(2, 0)));
		break;
	case Y:

		A[0] = cuCcos(hipCdiv(g.parameter, make_hipDoubleComplex(2, 0)));
		A[1] = cuCsin(hipCdiv(g.parameter, make_hipDoubleComplex(-2, 0)));
		A[2] = cuCsin(hipCdiv(g.parameter, make_hipDoubleComplex(-2, 0)));
		A[3] = cuCcos(g.parameter);
		break;

	case Z:

		A[0] = cuCexp(hipCmul(make_hipDoubleComplex(0,-1), hipCdiv(g.parameter, make_hipDoubleComplex(2, 0))));
		A[1] = { 0, 0 };
		A[2] = { 0, 0 };
		A[3] = cuCexp(hipCmul(make_hipDoubleComplex(0, -1), hipCdiv(g.parameter, make_hipDoubleComplex(2, 0))));
		break;
	}
	int i, j;
	int size1 = pow(2, index-1);
	for (i = 0; i <size1*size1; i++){
		*(B + i) = { 0, 0 };
	}
	for (i = 0; i <size1; i++){
		B[i*(size1) + i] = { 1, 0 };
	}

	Kronecker_CProduct(C, B, size1, size1, A, 2, 2);
	
	for (i = 0; i < 4 * size1 * size1; i++){
		A[i] = C[i];
	}
	int size2 = pow(2, (numberOfWires - index));
	for (i = 0; i <size2*size2; i++){
		*(B + i) = { 0, 0 };
	}

	for (i = 0; i <size2; i++){
		B[i*(size2 ) + i] = { 1, 0 };
	}
	Kronecker_CProduct(C, A, size1,size1, B, size2,size2);
	printm(C, size1*2*size2,size1*2*size2);

	
}
void generateInteractionMatrix(Gate g, hipDoubleComplex *A, hipDoubleComplex *B, hipDoubleComplex *C, int nrows, int ncols, int mrows, int mcols)
{

	int i;
	for (i = 0; i<nrows*ncols; i++){
		*(A + i) = {0,0};
	}
	A[0] = { 1, 0 };
	A[5] = cuCexp(hipCmul(g.parameter, {0,1}));
	A[10] = cuCexp(hipCmul(g.parameter, { 0, 1 }));
	A[15] = { 1, 0 };
	for (i = 0; i<nrows*ncols; i++){
		A[i] = hipCmul(A[i], cuCexp(hipCmul({ 0, -1 },hipCdiv(g.parameter, make_hipDoubleComplex(2, 0)))));
	}

	for (i = 0; i<mrows*mcols; i++){
		*(B + i) = { 0, 0 };
	}
	for (i = 0; i<mrows; i++){
		B[i*mcols + i] = { 1, 0 };
	}

	Kronecker_CProduct(C, A, nrows, ncols, B, mrows, mcols);
}

 
hipDoubleComplex* getGateMatrix(Gate g, int numberOfWires){
	int size = pow(2, numberOfWires);
	if (g.index1 == 0 && g.index2 == 0){
		
		hipDoubleComplex *C; // Resulting gate will be here
		C = new hipDoubleComplex[size*size];
		int i = 0;
		for (i = 0; i < size*size; i++){
			C[i] = { 0, 0 };
		}
		for (i = 0; i < size; i++){
			C[i * size + i] = { 1, 0 };
		}
		return C;
	}
	//same indexes mean this is an rotational gate
	if (g.index1 == g.index2){
		hipDoubleComplex *A = new hipDoubleComplex[size*size];
		hipDoubleComplex *B = new hipDoubleComplex[size*size];
		hipDoubleComplex *C = new hipDoubleComplex[size*size];
		int i = 0;
		for (i = 0; i < size*size; i++){
			C[i] = { 0, 0 };
		}
		for (i = 0; i < size; i++){
			C[i * size + i] = { 1, 0 };
		}
		// TODO optimize, g.index passed twice
		generateRotationalMatrix(g, A, B, C,g.index1,numberOfWires);

		return C;
	}
	//indexes differing by one mean it is interaction gate
	if (abs(g.index1 - g.index2) == 1){
		hipDoubleComplex *C; // Resulting gate will be here
		hipDoubleComplex *A; // Interaction between neighboring qbits
		int    nrows = 4;
		int    ncols = 4;
		hipDoubleComplex *B; // an Identity
		int    mrows = pow(2,size-2); //      The number of rows of B.                             //
		int    mcols = pow(2, size - 2); //     The number of cols of B.
		A = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex) * nrows * ncols);
		B = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex) * mrows * mcols);
		C = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex) * nrows * ncols * mrows * mcols);
		generateInteractionMatrix(g, A, B, C, nrows, ncols, mrows, mcols);
	/*	free(A);
		free(B);
		A = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex) * nrows * ncols * mrows * mcols);
		B = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex) * nrows * ncols * mrows * mcols);
		memcpy(A, C, sizeof(hipDoubleComplex) * nrows * ncols * mrows * mcols);
		memcpy(B, C, sizeof(hipDoubleComplex) * nrows * ncols * mrows * mcols);
		complexConj(B, nrows*mrows, ncols*mcols);
		transpose(B, nrows*mrows, ncols*mcols);
		multiplyC(C, A, nrows*mrows, ncols*mcols, B, nrows*mrows, ncols*mcols);
		printm(C, ncols*mcols, nrows*mrows);*/
		return C;
	}
	//indexes differing more than by one mean there should be swap gates
	if (abs(g.index1 - g.index2) > 1){
		//TODO implement swap and
	}

	return new hipDoubleComplex{ 0, 0 };
}
//TODO Consider Swap
Gate* generateChromosome(int sizeOfChromosome,int numberOfWires){
	int i = 0;
	int r;
	Gate *g = new	 Gate[sizeOfChromosome];
	
	srand(time(NULL));
	for (i = 0; i < sizeOfChromosome; i++){
		//r = 1+rand()%3; TODO uncomment
		r = 1;
		if (r == 1) {
			g[i].index1 = 1 + rand() % numberOfWires;
			g[i].index1 = 3;
			g[i].index2 = g[i].index1;
			g[i].direction = rand() % 3;
		}
		if (r == 2) {
			// TODO rewrite to universal
			g[i].index1 = 1;
			g[i].index2 = 2;
		}
		if (r == 3){
			g[i].index1 = 0;
			g[i].index2 = 0;
		}
		cout << g[i].index1;
		double parameter = 1.0;
			parameter *= rand() % 628 / 100;
			g[i].parameter = { parameter, 0 };
	}
	
	return g;
}
hipDoubleComplex* generateCircuitMatrix(Gate *g,int numberOfGates, int numberOfWires){
	int i;
	hipDoubleComplex *A, *B, *C;
	int size = pow(2, numberOfWires);
	A = new hipDoubleComplex[size*size];
	B = new hipDoubleComplex[size*size];
	C = new hipDoubleComplex[size*size];
	for (i = 0; i < size*size; i++){
		B[i] = { 0, 0 };
	}
	for (i = 0; i <size; i++){
		B[i * size + i] = { 1, 0 };
	}
	for (i = 0; i < numberOfGates; i++){
		C = getGateMatrix(g[i], numberOfWires);
		multiplyC(A, B, size, size, C, size, size);

		B = A;
	}
	return B;
}

Gate* mutation(Gate *g,int sizeOfChromosome,double probabilityOfMutation){
	int i = 0;
	int r;

	srand(time(NULL));
	for (i = 0; i < sizeOfChromosome; i++){
	//	r = 1 + rand() % 3; TODO uncomment
		r = 1;
		if (r == 1) {
			g[i].index1 = 1 + rand() % 2;
			g[i].index2 = g[i].index1;
			g[i].direction = rand() % 3;
		}
		if (r == 2) {
			// TODO rewrite to universal
			g[i].index1 = 1;
			g[i].index2 = 2;
		}
		if (r == 3){
			g[i].index1 = 0;
			g[i].index2 = 0;
		}
		double parameter = 1.0;
		parameter *= rand() % 628 / 100;
		g[i].parameter = { parameter, 0 };
	}

	return g;
}
int main(void) {
	
	Gate *chromosome = generateChromosome(1,4);// just initializes the chromosome
	//printm(getGateMatrix(*chromosome, 4), 8, 8);
	getGateMatrix(*chromosome, 3);
	// TODO consider passing by address
	//puts("!!!Hello World!!!"); /* prints !!!Hello World!!! */
	return EXIT_SUCCESS;
}
